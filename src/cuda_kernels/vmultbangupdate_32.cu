
#include <hip/hip_runtime.h>
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmultbang"
{
  __global__ void vmultbangupdate_32(const int lengthA, const float alpha, const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	  {
	    c[i] += alpha*a[i] * b[i];
	  }	
    }
}