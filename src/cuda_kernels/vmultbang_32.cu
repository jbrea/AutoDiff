
#include <hip/hip_runtime.h>
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C" 
{
  __global__ void vmultbang_32(const int lengthA, const float alpha, const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	  {
	    c[i] = alpha*a[i] * b[i];
	  }
    }
}